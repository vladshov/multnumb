// Test File read.cpp : Defines the entry point for the console application.
//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>

#include <math.h>
#include <ctime>

using namespace std;

int main (int argc, char *argv[])
{
	ifstream in_stream;
	in_stream.open(argv[1]);
	int a;
	int b;
	in_stream >> a;
	in_stream >> b;
	in_stream.close();
    int c;
	c = a * b;

	ofstream fs(argv[2]);

    if(!fs)
    {
        cerr<<"Cannot open the output file."<<endl;
        return 1;
    }
    fs<<c;
    fs.close();
	return 0;
}



